#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
//쿠다 프로그래밍을 위한 헤더파일
#include <iostream>
#include <ctime>
using namespace std;
void QuickSort(int a[], int l, int r);
void ShellSort(int a[], int n);
void oddeven(int a[], int n);
__global__ void odd(int* x, int n);
__global__ void even(int* x, int n);
void swap(int a[], int b, int c);
void checkSort(int a[], int n);

int main()
{
	cout << "12171856 주우민 첫번째 과제" << endl << endl;

	clock_t Quick_start, Quick_end;
	float Quick_res;

	clock_t OddCuda_start, OddCuda_end;
	float OddCuda_res;

	clock_t Shell_start, Shell_end;
	float Shell_res;
	//각각 quick sort, oddeven sort, shell sort의 시간을 측정하기 위한 변수들

	int arr1[5000];
	int temA1[5000];
	int temB1[5000];
	int temC1[5000];
	//첫번째는 5천개씩

	int arr2[50000];
	int temA2[50000];
	int temB2[50000];
	int temC2[50000];
	//두번째는 5만개씩

	int arr3[500000];
	int temA3[500000];
	int temB3[500000];
	int temC3[500000];
	//세번째는 50만개씩

	int arr[10];
	for (int i = 0; i < 10; i++)
	{
		arr[i] = 1;
	}
	
	OddCuda_start = clock();
	oddeven(arr, 10);
	OddCuda_end = clock();
	OddCuda_res = (float)(OddCuda_end - OddCuda_start);
	//실험해보니 어떠한 배열을 복사한 두배열을 oddeven 함수를 각각 돌려보면 첫번째 호출할때 비정상적으로 오래걸림 
	//즉main에서 cuda이용 함수를 처음 불러올때 시간이 더 소요된다 판단
	//정확한 시간계산을 위해 단순히 oddeven 한번 불러오는용
	

	srand(time(NULL));
	for (int i = 0; i < 5000; i++)
	{
		arr1[i] = rand() % 5000;
		temA1[i] = arr1[i];
		temB1[i] = arr1[i];
		temC1[i] = arr1[i];
	}
	srand(time(NULL));
	for (int i = 0; i < 50000; i++)
	{
		arr2[i] = rand() % 50000;
		temA2[i] = arr2[i];
		temB2[i] = arr2[i];
		temC2[i] = arr2[i];
	}
	srand(time(NULL));
	for (int i = 0; i < 500000; i++)
	{
		arr3[i] = rand() % 500000;
		temA3[i] = arr3[i];
		temB3[i] = arr3[i];
		temC3[i] = arr3[i];
	}
	//정확한 시간비교를 위해 5천, 5만, 50만개 크기의 배열에 랜덤으로 원소를 채워놓고 그거를 복사한 배열을 sort

	Quick_start = clock();
	QuickSort(temA1, 0, 4999);
	Quick_end = clock();
	Quick_res = (float)(Quick_end - Quick_start);
	cout << "Quick sort 5천개 소요 시간 = " << Quick_res << "ms" << endl;

	Shell_start = clock();
	ShellSort(temB1, 5000);
	Shell_end = clock();
	Shell_res = (float)(Shell_end - Shell_start);
	cout << "Shell sort 5천개 소요 시간 = " << Shell_res << "ms" << endl;

	OddCuda_start = clock();
	oddeven(temC1, 5000);
	OddCuda_end = clock();
	OddCuda_res = (float)(OddCuda_end - OddCuda_start);
	cout << "OddCuda sort 5천개 소요 시간 = " << OddCuda_res << "ms" << endl;
	//5천개짜리 배열 sort하고 시간 출력
	
	checkSort(temA1, 5000);
	checkSort(temB1, 5000);
	checkSort(temC1, 5000);
	cout << endl;
	//sort함수를 거친 배열이 정렬 됐는지 확인

	Quick_start = clock();
	QuickSort(temA2, 0, 49999);
	Quick_end = clock();
	Quick_res = (float)(Quick_end - Quick_start);
	cout << "Quick sort 5만개 소요 시간 = " << Quick_res << "ms" << endl;

	Shell_start = clock();
	ShellSort(temB2, 50000);
	Shell_end = clock();
	Shell_res = (float)(Shell_end - Shell_start);
	cout << "Shell sort 5만개 소요 시간 = " << Shell_res << "ms" << endl;

	OddCuda_start = clock();
	oddeven(temC2, 50000);
	OddCuda_end = clock();
	OddCuda_res = (float)(OddCuda_end - OddCuda_start);
	cout << "OddCuda sort 5만개 소요 시간 = " << OddCuda_res << "ms" << endl;

	checkSort(temA2, 50000);
	checkSort(temB2, 50000);
	checkSort(temC2, 50000);
	cout << endl;

	Quick_start = clock();
	QuickSort(temA3, 0, 499999);
	Quick_end = clock();
	Quick_res = (float)(Quick_end - Quick_start);
	cout << "Quick sort 5십만개 소요 시간 = " << Quick_res << "ms" << endl;

	Shell_start = clock();
	ShellSort(temB3, 500000);
	Shell_end = clock();
	Shell_res = (float)(Shell_end - Shell_start);
	cout << "Shell sort 5십만개 소요 시간 = " << Shell_res << "ms" << endl;
	
	OddCuda_start = clock();
	oddeven(temC3, 500000);
	OddCuda_end = clock();
	OddCuda_res = (float)(OddCuda_end - OddCuda_start);
	cout << "OddCuda sort 5십만개 소요 시간 = " << OddCuda_res << "ms" << endl;
	
	
	checkSort(temA3, 500000);
	checkSort(temB3, 500000);
	checkSort(temC3, 500000);
	//5만, 5십만 크기의 배열들도 동일하게 진행
}
void QuickSort(int a[], int l, int r)
//퀵소트 정의
{
	int i, j, m, v;
	if (r - l > 1)
		//쪼개진 배열안의 개수가 3개 이상이면
	{
		m = (l + r) / 2;
		//m은 l과 r의 중간값
		if (a[l] > a[m]) swap(a, l, m);
		if (a[l] > a[r]) swap(a, l, r);
		if (a[m] > a[r]) swap(a, m, r);
		//일단 첫번째, 중간번째, 마지막 숫자를 비교해서 정렬한다.
		swap(a, m, r - 1);
		//첫번째 중간번째 마지막 숫자는 정렬되어있으므로 m번째 값을 피봇으로 정할것이므로 피봇과 마지막 -1번째 숫자를 일단 바꿔주고
		v = a[r - 1]; i = l; j = r - 1;
		//밑에서 전위 연산자를 사용할것 이므로 i는첫번째 j는 피봇이랑 동일
		for (; ; ) {
			while (a[++i] < v);
			//while문의 첫번째 루프에서는 두번째값부터 피봇이랑 비교
			//피봇보다 큰 수를 스캔하면 while문 빠져나옴
			while (a[--j] > v);
			//while문의 첫번째 루프에서는 피봇 전값이랑 비교
			//피봇보다 작은 수를 스캔하면 while문 빠져나옴
			if (i >= j) break;
			//만약에 i>=j라면 즉 i와 j가 교차하면 피봇 자리를 찾은거니까 for문 빠져나옴
			swap(a, i, j);
			//교차하지 않았으면 스캔한 두수 바꾸기
		}
		swap(a, i, r - 1);
		//피봇 자리 i를 찾았으므로 피봇자리에 피봇을 위치시키고
		QuickSort(a, l, i - 1);
		//피봇 왼쪽을 퀵소트 돌리고
		QuickSort(a, i + 1, r);
		//피봇 오른쪽을 퀵소트 돌리고
	}
	else if (a[l] > a[r]) swap(a, l, r);
	//배열안의 개수가 2개 이하면 그냥 그 두수만 비교해서 스왑
}
void ShellSort(int a[], int n)
//쉘소트는 삽입정렬과 유사하다
//삽입정렬은 역순으로 정렬되어있을때 최악의 시간복잡도
//이를 보완하기 위해 간격을 두어 어느정도 정렬을 하고
//마지막에 gap을 1로두어 정렬을하면 어느정도 정렬이 된 배열을 정렬하는 삽입정렬이라 생각 가능
//따라서 삽입정렬보다 빠른 시간복잡도를 가짐
//여기서 사용한 방식은 A003462 방식으로 worst일떼 n^(3/2)시간복잡도를 가짐
{
	int k = 1, gap = 1;
	for (;;)
	{
		gap = pow(3, k);
		gap = (gap - 1) / 2;
		k++;
		if (gap > n / 3) break;
	}
	//A003462 방식으로 gap을 구한다
	for (; gap > 0; gap = gap / 3)
	//가장 큰 gap에서부터 시작해서 gap이 1이될때까지 for문은 돌아간다
	{
		for (int i = gap; i < n; i++)
		//gap만큼 떨어진 값들을 삽입정렬 하는 알고리즘
		//예를들어 n이 10이고 gap이 3이 되었다 치자
		//i가 3, 226번째 줄에서 3번 보관하고 for문에서 0번이랑 비교하고 둘이 정렬
		//i++ 쭉 되다 i가 6이 되면 226번째줄에서 6번 보관하고  3번이랑 비교, 3번이 더 크면 정렬하고 또 3번이랑 0번 비교하는 알고리즘
		//이렇게 0369 147 258 정렬됨
		{
			int j = 0;
			int temp = a[i];
			for (j = i; j >= gap && a[j - gap] > temp; j = j - gap)
			{
				a[j] = a[j - gap];
			}
			a[j] = temp;
		}
	}
}
void oddeven(int a[], int n)
//oddeven sort하는 함수
{
	int* d;
	//device에서 사용할 배열
	hipMalloc((void**)&d, n * sizeof(int));
	//device에 배열 크기만큼 메모리 할당
	hipMemcpy(d, a, n * sizeof(int), hipMemcpyHostToDevice);
	//gpu meemory에 배열 a를 d로 복사
	for (int i = 0; i <= n/2 ; i++)
	{
		even << <n/2, 1 >> > (d, n); //짝수
		//01 23 45 이런식으로 비교해서 정렬
		//비교하는 과정은 cuda를 이용 동시에 진행됨
		//<<<n/2,1>>> 의미는 동시에 n/2번의 함수가 호출된다는 뜻
		odd << <n/2, 1 >> > (d, n);  //홀수
		//위와 동일하게 12 34 56 이런식으로 비교해서 정렬

		//위 과정을 2/n번만큼 실행하면 끝
	}
	hipMemcpy(a, d, n * sizeof(int), hipMemcpyDeviceToHost);
	//결과를 cpu memory에 a로 복사
	hipFree(d);
	//해제
}
__global__ void odd(int* x, int n)
//쿠다 프로그래밍으로 구현한 odd even
//host: cpu, device: gpu
//cpu를 gpu로 옮기고 gpu가 처리한걸 다시 cpu로 복사하는 시스템
//__global__의 의미는 device에서 실행 host에서 호출
{
	int id = blockIdx.x;
	//블록 넘버를 id에 담고
	if (id * 2 + 2 < n)
	//n이 100이라 가정하면 12 34 56 78 비교하다 97 98 에서 끝나야됨
	{
		if (x[id * 2 +1] > x[id * 2 + 2])
		//12 34 56 ... 비교하고 정렬
		{
			int temp = x[id * 2 + 1];
			x[id * 2 +1] = x[id * 2 + 2];
			x[id * 2 + 2] = temp;
		}
	}
}
__global__ void even(int* x, int n)
//odd 함수와 동일한데 01 23 45... 이런식으로 비교
{
	int id = blockIdx.x;
	if (id * 2 + 1 < n)
	//n이 100이라 가정하면 01 23 45 비교하다 98 99 에서 끝나야됨
	{
		if (x[id * 2] > x[id * 2 + 1])
		{
			int temp = x[id * 2];
			x[id * 2] = x[id * 2 + 1];
			x[id * 2 + 1] = temp;
		}
	}
}
void swap(int a[], int b, int c)
//간단한 스왑함수
{
	int temp;
	temp = a[b];
	a[b] = a[c];
	a[c] = temp;
}
void checkSort(int a[], int n)
//정순 정렬이므로 체크소트는 단순히 배열 크기만큼 돌리면서 i번째 수가 i+1번째 수보다 작거나 같으면 sorted는 true 크면 false
//각각에 맞는 메세지 출력
{
	int i;
	bool sorted;
	sorted = true;
	for (i = 0; i < n - 1; i++) {
		if (a[i] > a[i + 1]) {
			sorted = false;
		}
		if (!sorted) {
			break;
		}
	}
	if (sorted) {
		printf("Sorting complete!\n");
	}
	else {
		printf("Error during sorting...\n");
	}
}
